// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "hipsparse.h"

int main() {
  printf("17. cuSPARSE API to hipSPARSE API synthetic test\n");

  // CHECK: hipsparseHandle_t handle_t;
  hipsparseHandle_t handle_t;

  // CHECK: hipsparseMatDescr_t matDescr_t, matDescr_t_2;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2;

  // CHECK: hipsparseColorInfo_t colorInfo_t;
  hipsparseColorInfo_t colorInfo_t;

  // CHECK: hipsparseOperation_t sparseOperation_t;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  hipsparseOperation_t sparseOperation_t;
  hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  // CHECK: hipsparseIndexBase_t indexBase_t;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;
  hipsparseIndexBase_t indexBase_t;
  hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;

  // CHECK: hipsparseMatrixType_t matrixType_t;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;
  hipsparseMatrixType_t matrixType_t;
  hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  // CHECK: hipsparseDiagType_t diagType_t;
  // CHECK-NEXT: hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  // CHECK-NEXT: hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;
  hipsparseDiagType_t diagType_t;
  hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;

  // CHECK: hipsparseFillMode_t fillMode_t;
  // CHECK-NEXT: hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  // CHECK-NEXT: hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;
  hipsparseFillMode_t fillMode_t;
  hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;

  // CHECK: hipsparseAction_t action_t;
  // CHECK-NEXT: hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  // CHECK-NEXT: hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;
  hipsparseAction_t action_t;
  hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;

  // CHECK: hipsparseDirection_t direction_t;
  // CHECK-NEXT: hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  // CHECK-NEXT: hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;
  hipsparseDirection_t direction_t;
  hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;

  // CHECK: hipsparseSolvePolicy_t solvePolicy_t;
  // CHECK-NEXT: hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  // CHECK-NEXT: hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
  hipsparseSolvePolicy_t solvePolicy_t;
  hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  // CHECK: hipsparsePointerMode_t pointerMode_t;
  // CHECK-NEXT: hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  // CHECK-NEXT: hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparsePointerMode_t pointerMode_t;
  hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;

  // CHECK: hipsparseStatus_t status_t;
  // CHECK-NEXT: hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  // CHECK-NEXT: hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT: hipsparseStatus_t STATUS_MAPPING_ERROR = HIPSPARSE_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipsparseStatus_t STATUS_EXECUTION_FAILED = HIPSPARSE_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsparseStatus_t STATUS_MATRIX_TYPE_NOT_SUPPORTED = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;
  hipsparseStatus_t status_t;
  hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  hipsparseStatus_t STATUS_MAPPING_ERROR = HIPSPARSE_STATUS_MAPPING_ERROR;
  hipsparseStatus_t STATUS_EXECUTION_FAILED = HIPSPARSE_STATUS_EXECUTION_FAILED;
  hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  hipsparseStatus_t STATUS_MATRIX_TYPE_NOT_SUPPORTED = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  int iVal = 0;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreate(cusparseHandle_t* handle);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreate(hipsparseHandle_t* handle);
  // CHECK: status_t = hipsparseCreate(&handle_t);
  status_t = hipsparseCreate(&handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroy(cusparseHandle_t handle);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroy(hipsparseHandle_t handle);
  // CHECK: status_t = hipsparseDestroy(handle_t);
  status_t = hipsparseDestroy(handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetStream(cusparseHandle_t handle, cudaStream_t streamId);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetStream(hipsparseHandle_t handle, hipStream_t streamId);
  // CHECK: status_t = hipsparseSetStream(handle_t, stream_t);
  status_t = hipsparseSetStream(handle_t, stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetStream(cusparseHandle_t handle, cudaStream_t* streamId);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetStream(hipsparseHandle_t handle, hipStream_t* streamId);
  // CHECK: status_t = hipsparseGetStream(handle_t, &stream_t);
  status_t = hipsparseGetStream(handle_t, &stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t mode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetPointerMode(hipsparseHandle_t handle, hipsparsePointerMode_t mode);
  // CHECK: status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);
  status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t* mode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetPointerMode(hipsparseHandle_t handle, hipsparsePointerMode_t* mode);
  // CHECK: status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);
  status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetVersion(cusparseHandle_t handle, int* version);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetVersion(hipsparseHandle_t handle, int* version);
  // CHECK: status_t = hipsparseGetVersion(handle_t, &iVal);
  status_t = hipsparseGetVersion(handle_t, &iVal);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateMatDescr(cusparseMatDescr_t* descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateMatDescr(hipsparseMatDescr_t* descrA);
  // CHECK: status_t = hipsparseCreateMatDescr(&matDescr_t);
  status_t = hipsparseCreateMatDescr(&matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyMatDescr(cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyMatDescr(hipsparseMatDescr_t descrA);
  // CHECK: status_t = hipsparseDestroyMatDescr(matDescr_t);
  status_t = hipsparseDestroyMatDescr(matDescr_t);

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 12000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCopyMatDescr(cusparseMatDescr_t dest, const cusparseMatDescr_t src);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCopyMatDescr(hipsparseMatDescr_t dest, const hipsparseMatDescr_t src);
  // CHECK: status_t = hipsparseCopyMatDescr(matDescr_t, matDescr_t_2);
  status_t = cusparseCopyMatDescr(matDescr_t, matDescr_t_2);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipsparseSpMatDescr_t spMatDescr_t;
  hipsparseSpMatDescr_t spMatDescr_t;

  // CHECK: hipsparseDnMatDescr_t dnMatDescr_t;
  hipsparseDnMatDescr_t dnMatDescr_t;

  // CHECK: hipsparseIndexType_t indexType_t;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;
  hipsparseIndexType_t indexType_t;
  hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;

  // CHECK: hipsparseFormat_t format_t;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;
  hipsparseFormat_t format_t;
  hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;

  // CHECK: hipsparseOrder_t order_t;
  // CHECK-NEXT: hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  // CHECK-NEXT: hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;
  hipsparseOrder_t order_t;
  hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;

  // CHECK: hipsparseSpMMAlg_t spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;
#endif

#if CUDA_VERSION >= 10010 && CUDA_VERSION < 12000
    // CHECK: hipsparseSpMMAlg_t COOMM_ALG1 = HIPSPARSE_COOMM_ALG1;
    // CHECK-NEXT: hipsparseSpMMAlg_t COOMM_ALG2 = HIPSPARSE_COOMM_ALG2;
    // CHECK-NEXT: hipsparseSpMMAlg_t COOMM_ALG3 = HIPSPARSE_COOMM_ALG3;
  cusparseSpMMAlg_t COOMM_ALG1 = CUSPARSE_COOMM_ALG1;
  cusparseSpMMAlg_t COOMM_ALG2 = CUSPARSE_COOMM_ALG2;
  cusparseSpMMAlg_t COOMM_ALG3 = CUSPARSE_COOMM_ALG3;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipsparseSpVecDescr_t spVecDescr_t;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: hipsparseDnVecDescr_t dnVecDescr_t;
  hipsparseDnVecDescr_t dnVecDescr_t;

  // CHECK: hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;
  hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;

  // CHECK: hipsparseSpMVAlg_t spMVAlg_t;
  hipsparseSpMVAlg_t spMVAlg_t;
#endif

#if CUDA_VERSION >= 10020 && CUDA_VERSION < 12000
  // CHECK: hipsparseFormat_t FORMAT_COO_AOS = HIPSPARSE_FORMAT_COO_AOS;
  cusparseFormat_t FORMAT_COO_AOS = CUSPARSE_FORMAT_COO_AOS;

  // CHECK: hipsparseSpMVAlg_t MV_ALG_DEFAULT = HIPSPARSE_MV_ALG_DEFAULT;
  cusparseSpMVAlg_t MV_ALG_DEFAULT = CUSPARSE_MV_ALG_DEFAULT;

  // CHECK: hipsparseSpMVAlg_t COOMV_ALG = HIPSPARSE_COOMV_ALG;
  // CHECK-NEXT: hipsparseSpMVAlg_t CSRMV_ALG1 = HIPSPARSE_CSRMV_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t CSRMV_ALG2 = HIPSPARSE_CSRMV_ALG2;
  cusparseSpMVAlg_t COOMV_ALG = CUSPARSE_COOMV_ALG;
  cusparseSpMVAlg_t CSRMV_ALG1 = CUSPARSE_CSRMV_ALG1;
  cusparseSpMVAlg_t CSRMV_ALG2 = CUSPARSE_CSRMV_ALG2;

  // CHECK: hipsparseSpMMAlg_t MM_ALG_DEFAULT = HIPSPARSE_MM_ALG_DEFAULT;
  // CHECK: hipsparseSpMMAlg_t CSRMM_ALG1 = HIPSPARSE_CSRMM_ALG1;
  cusparseSpMMAlg_t MM_ALG_DEFAULT = CUSPARSE_MM_ALG_DEFAULT;
  cusparseSpMMAlg_t CSRMM_ALG1 = CUSPARSE_CSRMM_ALG1;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipsparseStatus_t STATUS_INSUFFICIENT_RESOURCES = HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES;
  hipsparseStatus_t STATUS_INSUFFICIENT_RESOURCES = HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES;

  // CHECK: hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG4 = HIPSPARSE_SPMM_COO_ALG4;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;
  hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG4 = HIPSPARSE_SPMM_COO_ALG4;
  hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;

  // CHECK: hipsparseSpGEMMAlg_t spGEMMAlg_t;
  // CHECK-NEXT: hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;
  hipsparseSpGEMMAlg_t spGEMMAlg_t;
  hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;
#endif

#if CUDA_VERSION < 11000
  // CHECK: hipsparseHybMat_t hybMat_t;
  cusparseHybMat_t hybMat_t;

  // CHECK: hipsparseHybPartition_t hybPartition_t;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_AUTO = HIPSPARSE_HYB_PARTITION_AUTO;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_USER = HIPSPARSE_HYB_PARTITION_USER;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_MAX = HIPSPARSE_HYB_PARTITION_MAX;
  cusparseHybPartition_t hybPartition_t;
  cusparseHybPartition_t HYB_PARTITION_AUTO = CUSPARSE_HYB_PARTITION_AUTO;
  cusparseHybPartition_t HYB_PARTITION_USER = CUSPARSE_HYB_PARTITION_USER;
  cusparseHybPartition_t HYB_PARTITION_MAX = CUSPARSE_HYB_PARTITION_MAX;
#endif

#if CUDA_VERSION >= 11010
  // CHECK: hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  // CHECK-NEXT: hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;
  hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;

  // CHECK: hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  // CHECK-NEXT: hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
  hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;
  hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;

  // CHECK: hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;
  hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;

  // CHECK: hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;
  hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;

  // CHECK: hipsparseSDDMMAlg_t sDDMMAlg_t;
  // CHECK-NEXT: hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;
  hipsparseSDDMMAlg_t sDDMMAlg_t;
  hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;
#endif

#if CUDA_VERSION >= 11030
  // CHECK: hipsparseSpMatAttribute_t spMatAttribute_t;
  // CHECK-NEXT: hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  // CHECK-NEXT: hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;
  hipsparseSpMatAttribute_t spMatAttribute_t;
  hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;

  // CHECK: hipsparseSpSVAlg_t spSVAlg_t;
  // CHECK-NEXT: hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;
  hipsparseSpSVAlg_t spSVAlg_t;
  hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;

  // CHECK: hipsparseSpSMAlg_t spSMAlg_t;
  // CHECK-NEXT: hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;
  hipsparseSpSMAlg_t spSMAlg_t;
  hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;
#endif

  return 0;
}